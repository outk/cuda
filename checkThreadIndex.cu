#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call) {                                                       \
    const hipError_t error = call;                                         \
    if (error != hipSuccess) {                                             \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
}                                                                           

void printMatrix(int *C, const int nx, const int ny) {
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);
    for (int ix = 0; ix < nx; ix++) {
        printf("%3d", ic[ix]);
    }
    ic += nx;
    printf("\n");
    return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d) global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv) {
    printf("%s Starting ... \n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nx = 8;
    int ny = 6;
    int nxy = nx*ny;
    int nBytes = nxy * sizeof(float);

    int *h_A;
    h_A = (int *)malloc(nBytes);

    for (int i = 0; i < nxy; i++) {
        h_A[i] = i;
    }
    printMatrix(h_A, nx, ny);

    int *d_MatA;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));

    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));

    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
    CHECK(hipDeviceSynchronize());

    CHECK(hipFree(d_MatA));
    free(h_A);

    CHECK(hipDeviceReset());

    return(0);
}
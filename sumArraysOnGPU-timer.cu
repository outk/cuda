#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call) {                                                       \
    const hipError_t error = call;                                         \
    if (error != hipSuccess) {                                             \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
}     

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);
    
    double iStart, iElaps;

    iStart = cpuSecond();
    initialData (h_A, nElem);
    initialData (h_B, nElem);
    iElaps = cpuSecond() - iStart;
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    iStart = cpuSecond();
    sumArraysOnHost (h_A, h_B, hostRef, nElem);
    iElaps = cpuSecond() - iStart;

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));
    
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
    
    int iLen = 1024;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    iStart = cpuSecond();
    sumArraysOnGPU <<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnGPU <<<%d, %d>>> Time elapsed %f sec\n", grid.x, block.x, iElaps);

    CHECK(hipGetLastError());

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    checkResult(hostRef, gpuRef, nElem);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    
    return(0);
}